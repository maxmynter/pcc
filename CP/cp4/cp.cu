#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>

static inline void check(hipError_t err, const char *context) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error: " << context << ": " << hipGetErrorString(err)
              << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

static inline int divup(int a, int b) { return (a + b - 1) / b; }

#define CHECK(x) check(x, #x)

__global__ void compute_means_and_diffs(const float *data, float *diffs, int ny,
                                        int nx) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i >= ny)
    return;

  // Compute mean
  float mean = 0.0f;
  for (int k = 0; k < nx; k++) {
    mean += data[k + i * nx];
  }
  mean /= nx;

  // Compute differences and store them
  for (int k = 0; k < nx; k++) {
    diffs[k + i * nx] = data[k + i * nx] - mean;
  }
}

__global__ void compute_correlations(const float *diffs, float *result, int ny,
                                     int nx) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i >= ny || j > i)
    return;

  float sum_ij = 0.0f;
  float sum_i = 0.0f;
  float sum_j = 0.0f;

  for (int k = 0; k < nx; k++) {
    float diff_i = diffs[k + i * nx];
    float diff_j = diffs[k + j * nx];
    sum_ij += diff_i * diff_j;
    sum_i += diff_i * diff_i;
    sum_j += diff_j * diff_j;
  }

  float denominator = sqrt(sum_i * sum_j);
  result[i + j * ny] = denominator != 0.0f ? sum_ij / denominator : 0.0f;
}

__global__ void initialize_result(float *result, int ny) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < ny && j < ny) {
    result[i + j * ny] = 0.0f;
  }
}

void correlate(int ny, int nx, const float *data, float *result) {
  float *d_data = nullptr;
  float *d_diffs = nullptr;
  float *d_result = nullptr;

  // Allocate device memory
  CHECK(hipMalloc(&d_data, nx * ny * sizeof(float)));
  CHECK(hipMalloc(&d_diffs, nx * ny * sizeof(float)));
  CHECK(hipMalloc(&d_result, ny * ny * sizeof(float)));

  // Initialize result array with zeros
  dim3 init_block(16, 16);
  dim3 init_grid(divup(ny, 16), divup(ny, 16));
  initialize_result<<<init_grid, init_block>>>(d_result, ny);
  CHECK(hipGetLastError());

  // Copy input data to device
  CHECK(hipMemcpy(d_data, data, nx * ny * sizeof(float),
                   hipMemcpyHostToDevice));

  // Compute means and differences
  const int BLOCK_SIZE = 256;
  int grid_size = divup(ny, BLOCK_SIZE);
  compute_means_and_diffs<<<grid_size, BLOCK_SIZE>>>(d_data, d_diffs, ny, nx);
  CHECK(hipGetLastError());

  // Compute correlations
  dim3 corr_block(16, 16);
  dim3 corr_grid(divup(ny, 16), divup(ny, 16));
  compute_correlations<<<corr_grid, corr_block>>>(d_diffs, d_result, ny, nx);
  CHECK(hipGetLastError());
  CHECK(hipDeviceSynchronize());

  // Copy results back to host
  CHECK(hipMemcpy(result, d_result, ny * ny * sizeof(float),
                   hipMemcpyDeviceToHost));

  // Cleanup
  CHECK(hipFree(d_data));
  CHECK(hipFree(d_diffs));
  CHECK(hipFree(d_result));
}
